#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2016 by Contributors
 * \file multibox_prior.cu
 * \brief generate multibox prior boxes cuda kernels
 * \author Joshua Zhang
*/

#include "./multibox_prior-inl.h"
#include <mshadow/cuda/tensor_gpu-inl.cuh>

#define MULTIBOXPRIOR_CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    CHECK_EQ(error, hipSuccess) << " " << hipGetErrorString(error); \
  } while (0)

namespace mshadow {
namespace cuda {
template<typename DType>
__global__ void AssignPriors(DType *out, const float size, const float sqrt_ratio,
                             const int in_width, const int in_height,
                             const int img_width, const int img_height,
                             const float step_x, const float step_y,
                             const float center_offx, const float center_offy,
                             const int density_x, const int density_y,
                             const float dstep_x, const float dstep_y,
                             const int stride, const int offset) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= in_width * in_height) return;
  int r = index / in_width;
  int c = index % in_width;
  float center_x = (c + center_offx) * step_x;
  float center_y = (r + center_offy) * step_y;
  float w = size / img_width * sqrt_ratio / 2;  // half width
  float h = size / img_height / sqrt_ratio / 2;  // half height
  DType *ptr = out + index * stride + 4 * offset * density_x * density_y;
  for (int m = 0; m < density_x; ++m) {
    for (int n = 0; n < density_y; ++n) {
      *(ptr++) = center_x + dstep_x * step_x * (1 - density_x + 2 * m) - w;
      *(ptr++) = center_y + dstep_y * step_y * (1 - density_y + 2 * n) - h;
      *(ptr++) = center_x + dstep_x * step_x * (1 - density_x + 2 * m) + w;
      *(ptr++) = center_y + dstep_y * step_y * (1 - density_y + 2 * n) + h;
    }
  }
}
}  // namespace cuda

template<typename DType>
inline void MultiBoxPriorForward(const Tensor<gpu, 2, DType> &out,
                            const std::vector<float> &sizes,
                            const std::vector<float> &ratios,
                            const std::vector<int> &densities,
                            const int in_width, const int in_height,
                            const int img_width, const int img_height,
                            const std::vector<float> &steps,
                            const std::vector<float> &offsets) {
  CHECK_EQ(out.CheckContiguous(), true);
  hipStream_t stream = Stream<gpu>::GetStream(out.stream_);
  DType *out_ptr = out.dptr_;
  const float step_x = steps[1] > 0 ? steps[1] / img_width : 1.f / in_width;
  const float step_y = steps[0] > 0 ? steps[0] / img_height : 1.f / in_height;
  const int density_x = densities[1];
  const int density_y = densities[0];
  const float dstep_x = 1.f / (2 * density_x);
  const float dstep_y = 1.f / (2 * density_y);
  const float offset_x = offsets[1];
  const float offset_y = offsets[0];
  const int num_sizes = static_cast<int>(sizes.size());
  const int num_ratios = static_cast<int>(ratios.size());

  const int num_thread = cuda::kMaxThreadsPerBlock;
  dim3 dimBlock(num_thread);
  dim3 dimGrid((in_width * in_height - 1) / num_thread + 1);
  cuda::CheckLaunchParam(dimGrid, dimBlock, "MultiBoxPrior Forward");

  const int stride = 4 * (num_sizes + num_ratios - 1) * density_x * density_y;
  int offset = 0;
  // ratio = 1, various sizes
  for (int i = 0; i < num_sizes; ++i) {
    cuda::AssignPriors<DType><<<dimGrid, dimBlock, 0, stream>>>(out_ptr,
      sizes[i], 1.f, in_width, in_height, img_width, img_height, step_x, step_y,
      offset_x, offset_y, density_x, density_y, dstep_x, dstep_y, stride, offset);
    ++offset;
  }
  MULTIBOXPRIOR_CUDA_CHECK(hipPeekAtLastError());

  // size = sizes[0], various ratios
  for (int j = 1; j < num_ratios; ++j) {
    cuda::AssignPriors<DType><<<dimGrid, dimBlock, 0, stream>>>(out_ptr,
      sizes[0], sqrtf(ratios[j]), in_width, in_height, img_width, img_height, step_x, step_y,
       offset_x, offset_y, density_x, density_y, dstep_x, dstep_y, stride, offset);
    ++offset;
  }
  MULTIBOXPRIOR_CUDA_CHECK(hipPeekAtLastError());
}
}  // namespace mshadow

namespace mxnet {
namespace op {
template<>
Operator* CreateOp<gpu>(MultiBoxPriorParam param, int dtype) {
  Operator *op = NULL;
  MSHADOW_REAL_TYPE_SWITCH(dtype, DType, {
    op = new MultiBoxPriorOp<gpu, DType>(param);
  });
  return op;
}

}  // namespace op
}  // namespace mxnet
